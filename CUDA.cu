
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define SIZE  1024


__global__ void Add(int *c, int *a, int *b, int n){
    int i = threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

__global__ void Add_f(float *c, float *a, float *b, float n){
    int i = threadIdx.x;

    if (i < n) {
        c[i] = a[i] + b[i];
    }
}
/********************* calculate read and write bandwidth****************************/
int bandwidth(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    for (int i=1 ; i <= 1024*1024; i = i * 1024) {
        double size_0 = i;

        // Allocate memory block(in CPU), returns a pointer
        a = (int *)malloc(size_0*sizeof(int));
        b = (int *)malloc(size_0*sizeof(int));
        c = (int *)malloc(size_0*sizeof(int));

        // Allocate memory block(in GPU), returns a pointer
        hipMalloc( &d_a, size_0*sizeof(int));
        hipMalloc( &d_b, size_0*sizeof(int));
        hipMalloc( &d_c, size_0*sizeof(int));


        // structure for calculate the interval
        struct timeval start_read, end_read;
        struct timeval start_write, end_write;
        for (int j = 0; j < size_0; ++j) {
            a[j] = j;
            b[j] = j;
            c[j] = 0;
        }

        gettimeofday(&start_read, NULL);
	for (int i = 0; i < 1000000; i++) {
            hipMemcpy( d_a, a, size_0*sizeof(int), hipMemcpyHostToDevice ); // copy memory from CPU to GPU
            hipMemcpy( d_b, b, size_0*sizeof(int), hipMemcpyHostToDevice );
            hipMemcpy( d_c, c, size_0*sizeof(int), hipMemcpyHostToDevice );
	}
        gettimeofday(&end_read, NULL);

        // read_bandwidth = the size of memory copied from CPU to GPU /  interval
        float read_bandwidth = ((float)size_0*sizeof(int)*3*1000000)/(1000.0*(end_read.tv_sec-start_read.tv_sec)+(end_read.tv_usec-start_read.tv_usec)/1000.0);
	// (((size_0/(1024*1024))*3)*(1e6))/ (1000*1000*(end_read.tv_sec - start_read.tv_sec) + (end_read.tv_usec - start_read.tv_usec));
        printf ("with the size of %lf bytes, the read_bandwidth is %lf MBps\n ", size_0, read_bandwidth/1000.0);

        Add<<< 1, size_0 >>>(a, b, c, size_0);

        gettimeofday(&start_write, NULL);
	for (int i = 0; i < 1000000; i++) {
            hipMemcpy( c, d_c, size_0*sizeof(int), hipMemcpyDeviceToHost ); // copy memory from GPU to CPU
        }
	gettimeofday(&end_write, NULL);

         // read_bandwidth = the size of memory copied from GPU to CPU /  interval
        float write_bandwidth = ((float)size_0*sizeof(int)*1000000)/ ((float)(end_write.tv_sec - start_write.tv_sec)*1000.0*1000.0 +(float)(end_write.tv_usec - start_write.tv_usec));
        printf ("with the size of %lf bytes, the write_bandwidth is %lf MBps\n ", size_0, write_bandwidth);
    }

    // free memory

    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

/******************* claculate FLOPS and IOPS **********************/
int Operations(){
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    float *a_f, *b_f, *c_f;
    float *d_a_f, *d_b_f, *d_c_f;

    // declare interger and float variable(pointer) for calculation

    a = (int *)malloc(SIZE*sizeof(int));
    b = (int *)malloc(SIZE*sizeof(int));
    c = (int *)malloc(SIZE*sizeof(int));
    a_f = (float *)malloc(SIZE*sizeof(float));
    b_f = (float *)malloc(SIZE*sizeof(float));
    c_f = (float *)malloc(SIZE*sizeof(float));

    hipMalloc( &d_a, SIZE*sizeof(int));
    hipMalloc( &d_b, SIZE*sizeof(int));
    hipMalloc( &d_c, SIZE*sizeof(int));
    hipMalloc( &d_a_f, SIZE*sizeof(float));
    hipMalloc( &d_b_f, SIZE*sizeof(float));
    hipMalloc( &d_c_f, SIZE*sizeof(float));

    struct timeval start_seq, end_seq;
    struct timeval start_f, end_f;

    int i = 0;
    for (i = 0; i < SIZE; ++i) {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    for (i = 0; i < SIZE; ++i) {
        a_f[i] = i * 1.0;
        b_f[i] = i * 1.0;
        c_f[i] = 0.0;
    }


    hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( d_a_f, a_f, SIZE*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( d_b_f, b_f, SIZE*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( d_c_f, c_f, SIZE*sizeof(float), hipMemcpyHostToDevice );

    gettimeofday(&start_seq, NULL);


    int n = 0;
    // loop here is used to extend the operating time of CPU
    for (n = 0; n < 1000; n++) {
        Add<<< 1, SIZE >>>(a, b, c, SIZE); // calculate  a[ ] + b[ ] (interger)
    }

    gettimeofday(&end_seq, NULL);

    gettimeofday(&start_f, NULL);

    for (n = 0; n < 1000; n++) {
        Add_f<<< 1, SIZE >>>(a_f, b_f, c_f, SIZE); // calculate  a[ ] + b[ ] (float)
    }

    gettimeofday(&end_f, NULL);

    // IOPS = size * loop / interval
    float IOPS = ((SIZE*1000)/ ((1000.0 * (end_seq.tv_sec - start_seq.tv_sec) + (end_seq.tv_usec - start_seq.tv_usec) / 1000.0)/1000)/1e9);

    printf("the IOPS is %30f GIOPS\n", IOPS);

    // FLOPS = size * loop / interval
    float FLOPS = ((SIZE*1000)/ ((1000.0 * (end_f.tv_sec - start_f.tv_sec) + (end_f.tv_usec - start_f.tv_usec) / 1000.0)/1000)/1e9);

    printf("the FLOPS is %30f GFLOPS\n", FLOPS);

    //cudaMemcpy( c, d_c, SIZE*sizeof(int), cudaMemcpyDeviceToHost );

    //for (int i = 0; i < 10; ++i) {
        //printf("c[%d] = %d\n", i, c[i]);
    //}


    // free memory

    free(a);
    free(b);
    free(c);
    free(a_f);
    free(b_f);
    free(c_f);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    hipFree(d_a_f);
    hipFree(d_b_f);
    hipFree(d_c_f);

    return 0;
}

int main(void){
    bandwidth();
    Operations();
    return 0;
}
